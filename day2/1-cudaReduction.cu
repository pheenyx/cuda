
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include <getopt.h>
//
char* strInputFileName;
int N;
int isSerial = 0;
//
void checkCUDAError(const char *msg);
void parseArgs(int argc, char** argv);
void displayData(int* data, int size);
void loadData(int* data,char* fileName,int nElement);
void cudaFunction(int* inputData,int n);
void serialFunction(int* inputData,int n);
//example: the reduction kernel
__global__ void reduction(int* inputData, int n)
{
    //__global__ void reduce0(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = inputData[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) inputData[blockIdx.x] = sdata[0];
}

//}

int main(int argc, char** argv)
{
  int* inputData;
  //
  parseArgs(argc,argv);
  //
  inputData = (int*) malloc(sizeof(int)*N);
  //load n element from the input file
  loadData(inputData,strInputFileName,N);
  //display the input data, just use with the small data, to test
  printf("Input data:\n");
  displayData(inputData,N);
  printf("\n");
  //
  if(isSerial == 0)
  {
    printf("Running the CUDA implementation\n");
    cudaFunction(inputData,N);
    //
  }
  else
  {
    printf("Running the serial implementation\n");
    serialFunction(inputData,N);
  }
  //
  free(inputData);
  free(strInputFileName);
  //
  return 0;
}
//
void cudaFunction(int* inputData,int N)
{
  //the CUDA implementation here 
  int threadsPerBlock;
  int blocksPerGrid;
  int* device_input;
  int* host_output;
  hipEvent_t start, stop;
  float elapsedTime;
  unsigned int sharedSize;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //
  hipMalloc(&device_input, sizeof(int)*N); 
  host_output = (int*)malloc(sizeof(int)*N);
  //
  hipEventRecord(start,0);
  hipMemcpy(device_input,inputData,sizeof(int)*N,hipMemcpyHostToDevice); 
  checkCUDAError("cudaMemcpy: host to device");
  //
  sharedSize = N*sizeof(int);
  threadsPerBlock = N;
  blocksPerGrid = 1;
  reduction<<<blocksPerGrid,threadsPerBlock,sharedSize>>>(device_input, N); 
  hipDeviceSynchronize();
  checkCUDAError("kernel lauching");
  //use host_output to get the output from the kernel, 
  //the last element is the reduction result
  hipMemcpy(host_output,device_input,sizeof(int)*N,hipMemcpyDeviceToHost); 
  checkCUDAError("cudaMemcpy: device to host");
  //
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  printf("the reduction result is : %d\n",host_output[0]);
  printf("Elapsed time is: %f\n",elapsedTime);
  //
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
//
void serialFunction(int* inputData,int N)
{
  //the serial implementation here
  int sum =0;
  for (int i = 0; i < N; ++i){
    sum += inputData[i];
  }
  printf("the serial reduction result is : %d\n",sum);

}
//
void loadData(int* data,char* fileName,int nElement)
{
  FILE* fin;
  fin = fopen(fileName,"r");
  if(fin==NULL)
  {
    printf("Can not open %s\n",fileName);
    exit(1);
  }
  //
  fread(data,sizeof(int),nElement,fin);
  //
  fclose(fin);
}
//
void displayData(int* data, int size)
{
  int i;
  for(i=0;i<size;++i) printf("%d ",data[i]);
}
//
//function to check cuda error, cited from 
//http://www.drdobbs.com/parallel/cuda-supercomputing-for-the-masses-part/207603131?pgno=2
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
void parseArgs(int argc, char** argv)
{
  char c;
  int optionIndex = 0;
  struct option longOption[]=
  {
    {"input-file",1,NULL,'i'},
    {"number",1,NULL,'n'},
    {"is-serial",1,NULL,'s'},
    {0,0,0,0}
  };
  if (argc < 5) 
  {
    printf("Wrong number of arguments\n");
    exit(1);
  }
  while((c=getopt_long(argc,argv,"n:i:s",longOption,&optionIndex))!=-1)
  {
    switch(c)
    {
      case 'i':
	strInputFileName = strdup(optarg);
	break;
      case 'n':
	N = atoi(optarg);
	break;
      case 's':
	isSerial = 1;
	break;
      default:
	printf("Bad argument %c\n",c);
	exit(1);
    }
  }    
}

