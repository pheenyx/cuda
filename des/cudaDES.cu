#include "hip/hip_runtime.h"
#include "cudaDES.h"


// Regular implementation

unsigned char* plain;
unsigned char* cipher;
unsigned char* key;
int isSerial = 0;
int verbose = 0;

void des_key_set_parity( unsigned char key[DES_KEY_SIZE] )
{
    int i;

    for( i = 0; i < DES_KEY_SIZE; i++ )
        key[i] = odd_parity_table[key[i] / 2];
}

/*
 * Check the given key's parity, returns 1 on failure, 0 on SUCCESS
 */
int des_key_check_key_parity( const unsigned char key[DES_KEY_SIZE] )
{
    int i;

    for( i = 0; i < DES_KEY_SIZE; i++ )
        if ( key[i] != odd_parity_table[key[i] / 2] )
            return( 1 );

    return( 0 );
}


static void des_setkey( uint32_t SK[32], const unsigned char key[DES_KEY_SIZE] )
{
    int i;
    uint32_t X, Y, T;

    GET_UINT32_BE( X, key, 0 );
    GET_UINT32_BE( Y, key, 4 );

    /*
     * Permuted Choice 1
     */
    T =  ((Y >>  4) ^ X) & 0x0F0F0F0F;  X ^= T; Y ^= (T <<  4);
    T =  ((Y      ) ^ X) & 0x10101010;  X ^= T; Y ^= (T      );

    X =   (LHs[ (X      ) & 0xF] << 3) | (LHs[ (X >>  8) & 0xF ] << 2)
        | (LHs[ (X >> 16) & 0xF] << 1) | (LHs[ (X >> 24) & 0xF ]     )
        | (LHs[ (X >>  5) & 0xF] << 7) | (LHs[ (X >> 13) & 0xF ] << 6)
        | (LHs[ (X >> 21) & 0xF] << 5) | (LHs[ (X >> 29) & 0xF ] << 4);

    Y =   (RHs[ (Y >>  1) & 0xF] << 3) | (RHs[ (Y >>  9) & 0xF ] << 2)
        | (RHs[ (Y >> 17) & 0xF] << 1) | (RHs[ (Y >> 25) & 0xF ]     )
        | (RHs[ (Y >>  4) & 0xF] << 7) | (RHs[ (Y >> 12) & 0xF ] << 6)
        | (RHs[ (Y >> 20) & 0xF] << 5) | (RHs[ (Y >> 28) & 0xF ] << 4);

    X &= 0x0FFFFFFF;
    Y &= 0x0FFFFFFF;

    /*
     * calculate subkeys
     */
    for( i = 0; i < 16; i++ )
    {
        if( i < 2 || i == 8 || i == 15 )
        {
            X = ((X <<  1) | (X >> 27)) & 0x0FFFFFFF;
            Y = ((Y <<  1) | (Y >> 27)) & 0x0FFFFFFF;
        }
        else
        {
            X = ((X <<  2) | (X >> 26)) & 0x0FFFFFFF;
            Y = ((Y <<  2) | (Y >> 26)) & 0x0FFFFFFF;
        }

        *SK++ =   ((X <<  4) & 0x24000000) | ((X << 28) & 0x10000000)
                | ((X << 14) & 0x08000000) | ((X << 18) & 0x02080000)
                | ((X <<  6) & 0x01000000) | ((X <<  9) & 0x00200000)
                | ((X >>  1) & 0x00100000) | ((X << 10) & 0x00040000)
                | ((X <<  2) & 0x00020000) | ((X >> 10) & 0x00010000)
                | ((Y >> 13) & 0x00002000) | ((Y >>  4) & 0x00001000)
                | ((Y <<  6) & 0x00000800) | ((Y >>  1) & 0x00000400)
                | ((Y >> 14) & 0x00000200) | ((Y      ) & 0x00000100)
                | ((Y >>  5) & 0x00000020) | ((Y >> 10) & 0x00000010)
                | ((Y >>  3) & 0x00000008) | ((Y >> 18) & 0x00000004)
                | ((Y >> 26) & 0x00000002) | ((Y >> 24) & 0x00000001);

        *SK++ =   ((X << 15) & 0x20000000) | ((X << 17) & 0x10000000)
                | ((X << 10) & 0x08000000) | ((X << 22) & 0x04000000)
                | ((X >>  2) & 0x02000000) | ((X <<  1) & 0x01000000)
                | ((X << 16) & 0x00200000) | ((X << 11) & 0x00100000)
                | ((X <<  3) & 0x00080000) | ((X >>  6) & 0x00040000)
                | ((X << 15) & 0x00020000) | ((X >>  4) & 0x00010000)
                | ((Y >>  2) & 0x00002000) | ((Y <<  8) & 0x00001000)
                | ((Y >> 14) & 0x00000808) | ((Y >>  9) & 0x00000400)
                | ((Y      ) & 0x00000200) | ((Y <<  7) & 0x00000100)
                | ((Y >>  7) & 0x00000020) | ((Y >>  3) & 0x00000011)
                | ((Y <<  2) & 0x00000004) | ((Y >> 21) & 0x00000002);
    }
}

/*
 * DES key schedule (56-bit, encryption)
 */
int des_setkey_enc( des_context *ctx, const unsigned char key[DES_KEY_SIZE] )
{
    des_setkey( ctx->sk, key );

    return( 0 );
}

/*
 * DES key schedule (56-bit, decryption)
 */
int des_setkey_dec( des_context *ctx, const unsigned char key[DES_KEY_SIZE] )
{
    int i;

    des_setkey( ctx->sk, key );

    for( i = 0; i < 16; i += 2 )
    {
        SWAP( ctx->sk[i    ], ctx->sk[30 - i] );
        SWAP( ctx->sk[i + 1], ctx->sk[31 - i] );
    }

    return( 0 );
}


/*
 * DES-ECB block encryption/decryption
 */
int des_crypt_ecb( des_context *ctx,
                    const unsigned char input[8],
                    unsigned char output[8] )
{
    int i;
    uint32_t X, Y, T, *SK;

    SK = ctx->sk;

    GET_UINT32_BE( X, input, 0 );
    GET_UINT32_BE( Y, input, 4 );

    DES_IP( X, Y );

    for( i = 0; i < 8; i++ )
    {
        DES_ROUND( Y, X );
        DES_ROUND( X, Y );
    }

    DES_FP( Y, X );

    PUT_UINT32_BE( Y, output, 0 );
    PUT_UINT32_BE( X, output, 4 );

    return( 0 );
}


__global__ void DESkernel(volatile int* keyfound, unsigned char* key, const unsigned char* plain, const unsigned char* cipher, int size)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int inc = blockDim.x * gridDim.x; //#threads * #blocks
    int debug = 0;
    *keyfound = 0;
    
/*    printf("plain kernel\n");
    displayData_cuda(plain, size);
    printf("key kernel\n");
    displayData_cuda(key, size);
    printf("cipher kernel\n");
    displayData_cuda(cipher, size);
*/    
    des_context my_ctx;
    unsigned char buf[8];
    unsigned char my_key[8];
    memcpy(my_key,key,size);

    //initalize offset for threads
    newKey_cuda(my_key, tid);
    
    while(debug<500000 &&  !(*keyfound))
    {
/*        if ( tid == 0 && debug % 100 == 0){
            printf("debug %i!!! found: %i tid:%i my key:%c %02x   %c %02x   %c %02x   %c %02x   \n",debug,*keyfound,tid,my_key[0],my_key[0],my_key[1],my_key[1],my_key[2],my_key[2],my_key[3],my_key[3]);
        }
*/
        des_setkey_enc_cuda ( &my_ctx, my_key);
        //printf("found: %i tid:%i my key:%c %02x   %c %02x   %c %02x   %c %02x   \n",*keyfound,tid,my_key[0],my_key[0],my_key[1],my_key[1],my_key[2],my_key[2],my_key[3],my_key[3]);

        des_crypt_ecb_cuda( &my_ctx, plain, buf );
        //printf("tid:%i my cipher:%c %02x   %c %02x   %c %02x   %c %02x  \n",tid,buf[0],buf[0],buf[1],buf[1],buf[2],buf[2],buf[3],buf[3]);

        if (equals_cuda(buf, cipher))
        {
            printf("!!! KEY FOUND (tid %i, loops %i) !!!\n",tid, debug);
            printf("tid:%i key:%c %02X   %c %02X   %c %02X   %c %02X   %c %02X   %c %02X   %c %02X   %c %02X   \n",tid,my_key[0],my_key[0],my_key[1],my_key[1],my_key[2],my_key[2],my_key[3],my_key[3],my_key[4],my_key[4],my_key[5],my_key[5],my_key[6],my_key[6],my_key[7],my_key[7]);
            *keyfound = 1;
            memcpy(key, my_key, size);
            //break;
        }
        
        newKey_cuda(my_key, inc);
        ++debug;
        
    }
}

__device__ void newKey_cuda(unsigned char* key, int inc)
{
    *(uint64_t *)key += inc;
}

__device__ int equals_cuda(const unsigned char* a, const unsigned char* b)
{
    return (*(uint64_t*)a == *(uint64_t*)b);
}

__device__ int des_setkey_enc_cuda( des_context *ctx, const unsigned char key[DES_KEY_SIZE] )
{
    des_setkey_cuda( ctx->sk, key );
    return( 0 );
}
 
__device__ static void des_setkey_cuda( uint32_t SK[32], const unsigned char key[DES_KEY_SIZE] )
{
    int i;
    uint32_t X, Y, T;
    /*
     * PC1: left and right halves bit-swap
     */
    const uint32_t LHs[16] =
    {
        0x00000000, 0x00000001, 0x00000100, 0x00000101,
        0x00010000, 0x00010001, 0x00010100, 0x00010101,
        0x01000000, 0x01000001, 0x01000100, 0x01000101,
        0x01010000, 0x01010001, 0x01010100, 0x01010101
    };

    const uint32_t RHs[16] =
    {
        0x00000000, 0x01000000, 0x00010000, 0x01010000,
        0x00000100, 0x01000100, 0x00010100, 0x01010100,
        0x00000001, 0x01000001, 0x00010001, 0x01010001,
        0x00000101, 0x01000101, 0x00010101, 0x01010101,
    };


    GET_UINT32_BE( X, key, 0 );
    GET_UINT32_BE( Y, key, 4 );

    /*
     * Permuted Choice 1
     */
    T =  ((Y >>  4) ^ X) & 0x0F0F0F0F;  X ^= T; Y ^= (T <<  4);
    T =  ((Y      ) ^ X) & 0x10101010;  X ^= T; Y ^= (T      );

    X =   (LHs[ (X      ) & 0xF] << 3) | (LHs[ (X >>  8) & 0xF ] << 2)
        | (LHs[ (X >> 16) & 0xF] << 1) | (LHs[ (X >> 24) & 0xF ]     )
        | (LHs[ (X >>  5) & 0xF] << 7) | (LHs[ (X >> 13) & 0xF ] << 6)
        | (LHs[ (X >> 21) & 0xF] << 5) | (LHs[ (X >> 29) & 0xF ] << 4);

    Y =   (RHs[ (Y >>  1) & 0xF] << 3) | (RHs[ (Y >>  9) & 0xF ] << 2)
        | (RHs[ (Y >> 17) & 0xF] << 1) | (RHs[ (Y >> 25) & 0xF ]     )
        | (RHs[ (Y >>  4) & 0xF] << 7) | (RHs[ (Y >> 12) & 0xF ] << 6)
        | (RHs[ (Y >> 20) & 0xF] << 5) | (RHs[ (Y >> 28) & 0xF ] << 4);

    X &= 0x0FFFFFFF;
    Y &= 0x0FFFFFFF;

    /*
     * calculate subkeys
     */
    for( i = 0; i < 16; i++ )
    {
        if( i < 2 || i == 8 || i == 15 )
        {
            X = ((X <<  1) | (X >> 27)) & 0x0FFFFFFF;
            Y = ((Y <<  1) | (Y >> 27)) & 0x0FFFFFFF;
        }
        else
        {
            X = ((X <<  2) | (X >> 26)) & 0x0FFFFFFF;
            Y = ((Y <<  2) | (Y >> 26)) & 0x0FFFFFFF;
        }

        *SK++ =   ((X <<  4) & 0x24000000) | ((X << 28) & 0x10000000)
            | ((X << 14) & 0x08000000) | ((X << 18) & 0x02080000)
            | ((X <<  6) & 0x01000000) | ((X <<  9) & 0x00200000)
            | ((X >>  1) & 0x00100000) | ((X << 10) & 0x00040000)
            | ((X <<  2) & 0x00020000) | ((X >> 10) & 0x00010000)
            | ((Y >> 13) & 0x00002000) | ((Y >>  4) & 0x00001000)
            | ((Y <<  6) & 0x00000800) | ((Y >>  1) & 0x00000400)
            | ((Y >> 14) & 0x00000200) | ((Y      ) & 0x00000100)
            | ((Y >>  5) & 0x00000020) | ((Y >> 10) & 0x00000010)
            | ((Y >>  3) & 0x00000008) | ((Y >> 18) & 0x00000004)
            | ((Y >> 26) & 0x00000002) | ((Y >> 24) & 0x00000001);

        *SK++ =   ((X << 15) & 0x20000000) | ((X << 17) & 0x10000000)
            | ((X << 10) & 0x08000000) | ((X << 22) & 0x04000000)
            | ((X >>  2) & 0x02000000) | ((X <<  1) & 0x01000000)
            | ((X << 16) & 0x00200000) | ((X << 11) & 0x00100000)
            | ((X <<  3) & 0x00080000) | ((X >>  6) & 0x00040000)
            | ((X << 15) & 0x00020000) | ((X >>  4) & 0x00010000)
            | ((Y >>  2) & 0x00002000) | ((Y <<  8) & 0x00001000)
            | ((Y >> 14) & 0x00000808) | ((Y >>  9) & 0x00000400)
            | ((Y      ) & 0x00000200) | ((Y <<  7) & 0x00000100)
            | ((Y >>  7) & 0x00000020) | ((Y >>  3) & 0x00000011)
            | ((Y <<  2) & 0x00000004) | ((Y >> 21) & 0x00000002);
    }
}

__device__ int des_crypt_ecb_cuda( des_context *ctx,
        const unsigned char input[8],
        unsigned char output[8] )
{
    int i;
    uint32_t X, Y, T, *SK;
    const uint32_t SB1[64] =
    {
        0x01010400, 0x00000000, 0x00010000, 0x01010404,
        0x01010004, 0x00010404, 0x00000004, 0x00010000,
        0x00000400, 0x01010400, 0x01010404, 0x00000400,
        0x01000404, 0x01010004, 0x01000000, 0x00000004,
        0x00000404, 0x01000400, 0x01000400, 0x00010400,
        0x00010400, 0x01010000, 0x01010000, 0x01000404,
        0x00010004, 0x01000004, 0x01000004, 0x00010004,
        0x00000000, 0x00000404, 0x00010404, 0x01000000,
        0x00010000, 0x01010404, 0x00000004, 0x01010000,
        0x01010400, 0x01000000, 0x01000000, 0x00000400,
        0x01010004, 0x00010000, 0x00010400, 0x01000004,
        0x00000400, 0x00000004, 0x01000404, 0x00010404,
        0x01010404, 0x00010004, 0x01010000, 0x01000404,
        0x01000004, 0x00000404, 0x00010404, 0x01010400,
        0x00000404, 0x01000400, 0x01000400, 0x00000000,
        0x00010004, 0x00010400, 0x00000000, 0x01010004
    };

    const uint32_t SB2[64] =
    {
        0x80108020, 0x80008000, 0x00008000, 0x00108020,
        0x00100000, 0x00000020, 0x80100020, 0x80008020,
        0x80000020, 0x80108020, 0x80108000, 0x80000000,
        0x80008000, 0x00100000, 0x00000020, 0x80100020,
        0x00108000, 0x00100020, 0x80008020, 0x00000000,
        0x80000000, 0x00008000, 0x00108020, 0x80100000,
        0x00100020, 0x80000020, 0x00000000, 0x00108000,
        0x00008020, 0x80108000, 0x80100000, 0x00008020,
        0x00000000, 0x00108020, 0x80100020, 0x00100000,
        0x80008020, 0x80100000, 0x80108000, 0x00008000,
        0x80100000, 0x80008000, 0x00000020, 0x80108020,
        0x00108020, 0x00000020, 0x00008000, 0x80000000,
        0x00008020, 0x80108000, 0x00100000, 0x80000020,
        0x00100020, 0x80008020, 0x80000020, 0x00100020,
        0x00108000, 0x00000000, 0x80008000, 0x00008020,
        0x80000000, 0x80100020, 0x80108020, 0x00108000
    };

    const uint32_t SB3[64] =
    {
        0x00000208, 0x08020200, 0x00000000, 0x08020008,
        0x08000200, 0x00000000, 0x00020208, 0x08000200,
        0x00020008, 0x08000008, 0x08000008, 0x00020000,
        0x08020208, 0x00020008, 0x08020000, 0x00000208,
        0x08000000, 0x00000008, 0x08020200, 0x00000200,
        0x00020200, 0x08020000, 0x08020008, 0x00020208,
        0x08000208, 0x00020200, 0x00020000, 0x08000208,
        0x00000008, 0x08020208, 0x00000200, 0x08000000,
        0x08020200, 0x08000000, 0x00020008, 0x00000208,
        0x00020000, 0x08020200, 0x08000200, 0x00000000,
        0x00000200, 0x00020008, 0x08020208, 0x08000200,
        0x08000008, 0x00000200, 0x00000000, 0x08020008,
        0x08000208, 0x00020000, 0x08000000, 0x08020208,
        0x00000008, 0x00020208, 0x00020200, 0x08000008,
        0x08020000, 0x08000208, 0x00000208, 0x08020000,
        0x00020208, 0x00000008, 0x08020008, 0x00020200
    };

    const uint32_t SB4[64] =
    {
        0x00802001, 0x00002081, 0x00002081, 0x00000080,
        0x00802080, 0x00800081, 0x00800001, 0x00002001,
        0x00000000, 0x00802000, 0x00802000, 0x00802081,
        0x00000081, 0x00000000, 0x00800080, 0x00800001,
        0x00000001, 0x00002000, 0x00800000, 0x00802001,
        0x00000080, 0x00800000, 0x00002001, 0x00002080,
        0x00800081, 0x00000001, 0x00002080, 0x00800080,
        0x00002000, 0x00802080, 0x00802081, 0x00000081,
        0x00800080, 0x00800001, 0x00802000, 0x00802081,
        0x00000081, 0x00000000, 0x00000000, 0x00802000,
        0x00002080, 0x00800080, 0x00800081, 0x00000001,
        0x00802001, 0x00002081, 0x00002081, 0x00000080,
        0x00802081, 0x00000081, 0x00000001, 0x00002000,
        0x00800001, 0x00002001, 0x00802080, 0x00800081,
        0x00002001, 0x00002080, 0x00800000, 0x00802001,
        0x00000080, 0x00800000, 0x00002000, 0x00802080
    };

    const uint32_t SB5[64] =
    {
        0x00000100, 0x02080100, 0x02080000, 0x42000100,
        0x00080000, 0x00000100, 0x40000000, 0x02080000,
        0x40080100, 0x00080000, 0x02000100, 0x40080100,
        0x42000100, 0x42080000, 0x00080100, 0x40000000,
        0x02000000, 0x40080000, 0x40080000, 0x00000000,
        0x40000100, 0x42080100, 0x42080100, 0x02000100,
        0x42080000, 0x40000100, 0x00000000, 0x42000000,
        0x02080100, 0x02000000, 0x42000000, 0x00080100,
        0x00080000, 0x42000100, 0x00000100, 0x02000000,
        0x40000000, 0x02080000, 0x42000100, 0x40080100,
        0x02000100, 0x40000000, 0x42080000, 0x02080100,
        0x40080100, 0x00000100, 0x02000000, 0x42080000,
        0x42080100, 0x00080100, 0x42000000, 0x42080100,
        0x02080000, 0x00000000, 0x40080000, 0x42000000,
        0x00080100, 0x02000100, 0x40000100, 0x00080000,
        0x00000000, 0x40080000, 0x02080100, 0x40000100
    };

    const uint32_t SB6[64] =
    {
        0x20000010, 0x20400000, 0x00004000, 0x20404010,
        0x20400000, 0x00000010, 0x20404010, 0x00400000,
        0x20004000, 0x00404010, 0x00400000, 0x20000010,
        0x00400010, 0x20004000, 0x20000000, 0x00004010,
        0x00000000, 0x00400010, 0x20004010, 0x00004000,
        0x00404000, 0x20004010, 0x00000010, 0x20400010,
        0x20400010, 0x00000000, 0x00404010, 0x20404000,
        0x00004010, 0x00404000, 0x20404000, 0x20000000,
        0x20004000, 0x00000010, 0x20400010, 0x00404000,
        0x20404010, 0x00400000, 0x00004010, 0x20000010,
        0x00400000, 0x20004000, 0x20000000, 0x00004010,
        0x20000010, 0x20404010, 0x00404000, 0x20400000,
        0x00404010, 0x20404000, 0x00000000, 0x20400010,
        0x00000010, 0x00004000, 0x20400000, 0x00404010,
        0x00004000, 0x00400010, 0x20004010, 0x00000000,
        0x20404000, 0x20000000, 0x00400010, 0x20004010
    };

    const uint32_t SB7[64] =
    {
        0x00200000, 0x04200002, 0x04000802, 0x00000000,
        0x00000800, 0x04000802, 0x00200802, 0x04200800,
        0x04200802, 0x00200000, 0x00000000, 0x04000002,
        0x00000002, 0x04000000, 0x04200002, 0x00000802,
        0x04000800, 0x00200802, 0x00200002, 0x04000800,
        0x04000002, 0x04200000, 0x04200800, 0x00200002,
        0x04200000, 0x00000800, 0x00000802, 0x04200802,
        0x00200800, 0x00000002, 0x04000000, 0x00200800,
        0x04000000, 0x00200800, 0x00200000, 0x04000802,
        0x04000802, 0x04200002, 0x04200002, 0x00000002,
        0x00200002, 0x04000000, 0x04000800, 0x00200000,
        0x04200800, 0x00000802, 0x00200802, 0x04200800,
        0x00000802, 0x04000002, 0x04200802, 0x04200000,
        0x00200800, 0x00000000, 0x00000002, 0x04200802,
        0x00000000, 0x00200802, 0x04200000, 0x00000800,
        0x04000002, 0x04000800, 0x00000800, 0x00200002
    };

    const uint32_t SB8[64] =
    {
        0x10001040, 0x00001000, 0x00040000, 0x10041040,
        0x10000000, 0x10001040, 0x00000040, 0x10000000,
        0x00040040, 0x10040000, 0x10041040, 0x00041000,
        0x10041000, 0x00041040, 0x00001000, 0x00000040,
        0x10040000, 0x10000040, 0x10001000, 0x00001040,
        0x00041000, 0x00040040, 0x10040040, 0x10041000,
        0x00001040, 0x00000000, 0x00000000, 0x10040040,
        0x10000040, 0x10001000, 0x00041040, 0x00040000,
        0x00041040, 0x00040000, 0x10041000, 0x00001000,
        0x00000040, 0x10040040, 0x00001000, 0x00041040,
        0x10001000, 0x00000040, 0x10000040, 0x10040000,
        0x10040040, 0x10000000, 0x00040000, 0x10001040,
        0x00000000, 0x10041040, 0x00040040, 0x10000040,
        0x10040000, 0x10001000, 0x10001040, 0x00000000,
        0x10041040, 0x00041000, 0x00041000, 0x00001040,
        0x00001040, 0x00040040, 0x10000000, 0x10041000
    };

    SK = ctx->sk;

    GET_UINT32_BE( X, input, 0 );
    GET_UINT32_BE( Y, input, 4 );

    DES_IP( X, Y );

    for( i = 0; i < 8; i++ )
    {
        DES_ROUND( Y, X );
        DES_ROUND( X, Y );
    }

    DES_FP( Y, X );

    PUT_UINT32_BE( Y, output, 0 );
    PUT_UINT32_BE( X, output, 4 );

    return( 0 );
}

__device__ void displayData_cuda(const unsigned char* data, int size)
{
    for (int i = 0; i<size; ++i){
        printf("%c %02x\t",data[i],data[i]);
    }
    printf("\n");
}


void cudaFunction(unsigned char* key, const unsigned char* plain, const unsigned char* cipher, int size)
{   
    unsigned char* startkey = (unsigned char*)malloc(sizeof(unsigned char)*size);
    memcpy(startkey,key,size);

    hipEvent_t start, stop;
    float elapsedTime;
    size_t real_size;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    real_size = size * sizeof(unsigned char);
    unsigned char* d_key;
    unsigned char* d_plain;
    unsigned char* d_cipher;
    int* d_keyfound;

    //malloc device memory
    hipMalloc(&d_key, real_size);
    checkCUDAError("hipMalloc d_key");
    hipMalloc(&d_plain, real_size);
    checkCUDAError("hipMalloc d_plain");
    hipMalloc(&d_cipher, real_size);
    checkCUDAError("hipMalloc d_cipher");
    hipMalloc(&d_keyfound, sizeof(int));
    checkCUDAError("hipMalloc d_keyfound");

    //copy to device
    hipMemcpy(d_key, key, real_size, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy to device key");
    hipMemcpy(d_plain, plain, real_size, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy to device plain");
    hipMemcpy(d_cipher, cipher, real_size, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy to device cipher");

    
    //invoke kernel
    int numberBlocks = 64;     //64;
    int numberThreads = 32;    //32;
    //nt sharedSize = 3*real_size+sizeof(des_context);
    DESkernel<<<numberBlocks, numberThreads>>>(d_keyfound, d_key, d_plain, d_cipher, size);
    checkCUDAError("cudakernel call");
    
    //copy back to host
    hipMemcpy(key, d_key, real_size, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy to horst cipher");

    //stop recorder and print time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("the result is :\n");
    if (equals(startkey,key)){
        printf("Key was not found\n");
    } else {
        displayData(key, size);
    }
    printf("Elapsed time is: %f\n",elapsedTime);
    //
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Free device memory
    hipFree(d_key);
    hipFree(d_plain);
    hipFree(d_cipher);
    hipFree(d_keyfound);
}

/*
 * Main routine
 */
int main( int argc, char** argv )
{
    des_context my_ctx;
    unsigned char buf[8];



/*    static unsigned char my_keys[8] =
    {
        0x60, 0x65, 0x79, 0x69, 0x65, 0x79, 0x6B, 0x65
    };
    static const unsigned char my_keys[24] =
    {
        0x6B, 0x65, 0x79, 0x6B, 0x65, 0x79, 0x6B, 0x65,
        0x23, 0x45, 0x67, 0x89, 0xAB, 0xCD, 0xEF, 0x01,
        0x45, 0x67, 0x89, 0xAB, 0xCD, 0xEF, 0x01, 0x23
    };

    static const unsigned char my_plain[3][8] =
    {
        { 0x70, 0x6C, 0x61, 0x69, 0x6E, 0x31, 0x32, 0x33 },
        { 0x70, 0x6C, 0x61, 0x69, 0x6E, 0x34, 0x35, 0x36 },
        { 0x70, 0x6C, 0x61, 0x69, 0x6E, 0x37, 0x38, 0x39 }
    }; 
    
    static unsigned char my_cipher[8] =
    {
        0x1B, 0xCD, 0xB8, 0x89, 0x88, 0xE2, 0x02, 0x7F
    };   
*/
    printf("\n");


    parseArgs(argc,argv);


    if (verbose) {
        printf("start key:\n");
        displayData(key, 8);
        printf("plain:\n");
        displayData(plain, 8);
        printf("cipher:\n");
        displayData(cipher, 8);
    }
    //
    if(isSerial == 0)
    {
        printf("Running the CUDA implementation\n");
        cudaFunction(key,plain,cipher,8);
        //
    }
    else
    {
        printf("Running the serial implementation\n");
        hipEvent_t start, stop;
        float elapsedTime;
        int size = 8;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start,0);
        

        printf("=====START======\n");
        
        
        int keyfound = 0;
        long i = 0;

        printf("plain cpu\n");
        displayData(plain, size);
        printf("start key cpu\n");
        displayData(key, size);
        unsigned char my_key[8];
        memcpy(my_key,key,size);
        unsigned char found_key[8];
        memcpy(found_key, key, size);

        while(i<500000000 && !(keyfound))
        {
/*            if ( i % 100000 == 0){
                printf("loop %i!!! found: %i my key:%c %02x   %c %02x   %c %02x   %c %02x   \n",i,keyfound,my_key[0],my_key[0],my_key[1],my_key[1],my_key[2],my_key[2],my_key[3],my_key[3]);
            }
*/
            des_setkey_enc ( &my_ctx, my_key);


            des_crypt_ecb ( &my_ctx, plain, buf );

            if (equals(buf, cipher))
            {
                printf("!!! KEY FOUND (loop %li)!!!\n",i);
                keyfound = 1;
                memcpy(found_key, my_key, size);
                break;
            }

            newKey(my_key);
            ++i;

        }

        printf("=====END========\n");

        printf("\n");

        //stop recorder and print time
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime,start,stop);
        printf("the result is :\n");
        if (equals(key,found_key)){
            printf("Key was not found\n");
        } else {
            displayData(my_key, size);
        }
        printf("Elapsed time is: %f\n",elapsedTime);
        //
        hipEventDestroy(start);
        hipEventDestroy(stop);

    }
    return ( 0 );
}

void newKey(unsigned char* key)
{
    ++*(uint64_t *)key;
}

int equals(unsigned char* a, unsigned char* b)
{
    return (*(uint64_t*)a == *(uint64_t*)b);
}

void displayData(const unsigned char* data, int size)
{
    for (int i = 0; i<size; ++i){
        printf("%c %02x\t",data[i],data[i]);
    }
    printf("\n");
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

unsigned char* convert(char *s)
{
    unsigned char* val = (unsigned char*) malloc(strlen(s)/2);
    /* WARNING: no sanitization or error-checking whatsoever */
    for(int count = 0; count < sizeof(val)/sizeof(val[0]); count++) {
        sscanf(s, "%2hhx", &val[count]);
        s += 2 * sizeof(char);
    }
    return val;
}


void parseArgs(int argc, char** argv)
{
    char c;
    char* cipherIn;
    char* keyIn;
    int optionIndex = 0;
    struct option longOption[]=
    {
        {"plaintext",1,NULL,'p'},
        {"ciphertext",1,NULL,'c'},
        {"startkey",1,NULL,'k'},
        {"serial",1,NULL,'s'},
        {"verbose",1,NULL,'v'},
        {0,0,0,0}
    };
    if (argc < 6) 
    {
        printf("Wrong number of arguments\n");
        exit(1);
    }
    while((c=getopt_long(argc,argv,"p:c:k:sv",longOption,&optionIndex))!=-1)
    {
        switch(c)
        {
            case 'p':
                plain = (unsigned char*)strdup(optarg);
                break;
            case 'c':
                cipherIn = strdup(optarg);
                cipher = convert(cipherIn);
                break;
            case 'k':
                keyIn = strdup(optarg);
                key = convert(keyIn);
                break;
            case 's':
                isSerial = 1;
                break;
            case 'v':
                verbose = 1;
                break;
            default:
                printf("Bad argument %c\n",c);
                exit(1);
        }
    }    
}

